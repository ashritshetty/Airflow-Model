#include "hip/hip_runtime.h"
__global__ void cascade(float* to, float* from, unsigned char *ncls)
{

	int i, j, k;
	i = get_global_id(0);
	j = get_global_id(1);
	k = get_global_id(2);

	if(ncls[cstore(i,j,k)] == FREE || ncls[cstore(i,j,k)] == FFLOW){
		struct rvector uptr;
		double rhoptr;
		get_rho_u(from, i, j, k, &rhoptr, &uptr);
		for(int l=0;l<DIRECTIONS;++l){
			{

				float equi=get_equilibrium(rhoptr,uptr, l);

				float curr_value= from[store(i,j,k,l)];
				float sub =  omega* (curr_value - equi );
				to[store(i,j,k,l)] = curr_value - sub ;

			}

		}


	}


//printf("I am in cascade");
return;
}
